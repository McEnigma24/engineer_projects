#include "RT_Renderer.h"
#include "CTRL_Setuper.h"


// CTRL     - Constrol sofware - stast, execution, changing scenes
// RT       - Ray Tracing related structures and functions
// FUNC     - Generic Math


void good_to_go(int argc, char* argv[])
{
    if(argc >= 4)
    {
        CPU_LINE(G::MODEL_NAME = argv[1]);
        GPU_LINE(G::MODEL_NAME = argv[2]);

        CPU_LINE(G::NUM_OF_CORES = argv[3]);
        GPU_LINE(G::NUM_OF_CORES = "0");

        if(argc == 6)
        {
            G::SCALING_MULTI = atof(argv[4]); 
            G::SCALING_ADD = atof(argv[5]); 
        }
    }
    else
    {
        cout << "NOT ENOUGHT PARAMS fead into main function\n"; 
        exit(0);
    }

    #ifdef GPU
        string no_gpu = "nvidia_gpu_not_present";
        if(G::MODEL_NAME == no_gpu)
        {
            cout << "Nvidia GPU not present. Exiting...\n";
            exit(0);
        }
    #endif

    ifstream FILE(path_rep_num);
    if (!FILE.is_open()) {
        std::cerr << "Nie mozna otworzyc pliku: " << path_rep_num << std::endl;
        exit(0);
    }
    FILE >> G::REP_NUMBER;

    if(G::REP_NUMBER == 0) { cout << "REP_NUMBER == 0\n"; exit(0); }
}

void macros_check()
{
    // WIN & LINUX
    #if !defined(WIN) && !defined(LINUX)
    line("ERROR - WIN - LINUX - NONE of the platform macros were defined");
    exit(0);
    #endif
    #if defined(WIN) && defined(LINUX)
    line("ERROR - WIN - LINUX - BOTH of the platform macros were defined");
    exit(0);
    #endif

    // CPU & GPU
    #if !defined(CPU) && !defined(GPU)
    line("ERROR - CPU - GPU - NONE of the platform macros were defined");
    exit(0);
    #endif
    #if defined(CPU) && defined(GPU)
    line("ERROR - CPU - GPU - BOTH of the platform macros were defined");
    exit(0);
    #endif

    #ifdef CPU
    // ARCH_X86 & ARCH_X64
    #if !defined(ARCH_X86) && !defined(ARCH_X64)
    line("ERROR - ARCH_X86 - ARCH_X64 - NONE of the platform macros were defined");
    exit(0);
    #endif
    #if defined(ARCH_X86) && defined(ARCH_X64)
    line("ERROR - ARCH_X86 - ARCH_X64 - BOTH of the platform macros were defined");
    exit(0);
    #endif
    #endif

    // UNIT_FLOAT & UNIT_DOUBLE
    #if !defined(UNIT_FLOAT) && !defined(UNIT_DOUBLE)
    line("ERROR - UNIT_FLOAT - UNIT_DOUBLE - NONE of the platform macros were defined");
    exit(0);
    #endif
    #if defined(UNIT_FLOAT) && defined(UNIT_DOUBLE)
    line("ERROR - UNIT_FLOAT - UNIT_DOUBLE - BOTH of the platform macros were defined");
    exit(0);
    #endif
}

#define exiting_app { WIN_LINE(window.close()); return 0; }
bool display_controller = false;

int main(int argc, char* argv[])
{
    good_to_go(argc, argv);
    macros_check();

    srand((unsigned int)time(NULL));
    Setuper::setup_Global_Variables___and___Clear_Stats();

    #ifdef LINUX
        Renderer render;

        for (;;)
        {
            render.RENDER();

            if (render.test_is_finished()) { exiting_app; }
        }

    #endif

    #ifdef WIN
        sf::RenderWindow window(sf::VideoMode(static_cast<unsigned int>(G::WIDTH), static_cast<unsigned int>(G::HEIGHT)), "Ray Tracer");
        Renderer render(window);

        #ifdef INTERACTIVE_MODE
        sf::Mouse::setPosition(Vector2i(G::WIDTH / 2, G::HEIGHT / 2), window);
        #endif

        while (window.isOpen())
        {
            sf::Event event;
            while (window.pollEvent(event))
            {
                switch (event.type)
                {
                case sf::Event::Closed:
                {
                    exiting_app;
                }

                case sf::Event::MouseWheelMoved:
                {
                    G::Camera::pos.z += u(event.mouseWheel.delta * G::Camera::move_speed);

                    /*if(event.mouseWheel.delta > 0)
                        G::Camera::rotation_angles.x -= G::Camera::rotation_speed;
                    else
                        G::Camera::rotation_angles.x += G::Camera::rotation_speed;*/



                    break;
                }

                #ifdef INTERACTIVE_MODE
                case sf::Event::MouseMoved:
                {
                    if (event.mouseMove.x == G::WIDTH / 2 && event.mouseMove.y == G::HEIGHT / 2) break;

                    if (G::WIDTH / 2 < event.mouseMove.x)
                    {
                        G::Camera::rotation_angles.x -= G::Camera::rotation_speed;
                    }
                    else
                    {
                        G::Camera::rotation_angles.x += G::Camera::rotation_speed;
                    }

                    if (G::HEIGHT / 2 < event.mouseMove.y)
                    {
                        G::Camera::rotation_angles.y -= G::Camera::rotation_speed;
                    }
                    else
                    {
                        G::Camera::rotation_angles.y += G::Camera::rotation_speed;
                    }

                    sf::Mouse::setPosition(Vector2i(G::WIDTH / 2, G::HEIGHT / 2), window);
                    break;
                }
                #endif // INTERACTIVE_MODE

                case sf::Event::MouseButtonPressed:
                {
                    if (sf::Mouse::isButtonPressed(sf::Mouse::Left))
                    {
                        // cmp = sf::Mouse::getPosition(window); //cout << cmp.x << ", " << cmp.y << endl;

                        // buttons.click_check_buttons(cmp);
                    }

                    break;
                }

                case sf::Event::KeyPressed:
                {
                    switch (event.key.code)
                    {
                    case sf::Keyboard::Escape:
                    {
                        exiting_app;
                    }

                    case sf::Keyboard::Num1:
                    {
                        G::PIXEL_SKIP = 1;
                        render.reset_all_pixels();
                        break;
                    }
                    case sf::Keyboard::Num2:
                    {
                        G::PIXEL_SKIP = 2;
                        render.reset_all_pixels();
                        break;
                    }
                    case sf::Keyboard::Num3:
                    {
                        G::PIXEL_SKIP = 3;
                        render.reset_all_pixels();
                        break;
                    }

                    case sf::Keyboard::Numpad0:
                    {
                        display_controller = !display_controller;
                        delay_input;
                        break;
                    }

                    #ifdef INTERACTIVE_MODE
                    case sf::Keyboard::Q:
                    {
                        G::Camera::rotation_angles.z += G::Camera::rotation_speed * 10;
                        break;
                    }

                    case sf::Keyboard::E:
                    {
                        G::Camera::rotation_angles.z -= G::Camera::rotation_speed * 10;
                        break;
                    }

                    case sf::Keyboard::A:
                    {
                        G::Camera::move_offset.x -= G::Camera::move_speed;





                        // Zrobi�, �eby w zale�no�ci od rotation_angles mo�na by�o si� porusza� 
                        // po x, y, z  np. id�c tylko do przodu





                        break;
                    }

                    case sf::Keyboard::D:
                    {
                        G::Camera::move_offset.x += G::Camera::move_speed;
                        break;
                    }

                    case sf::Keyboard::W:
                    {
                        G::Camera::move_offset.z += G::Camera::move_speed * 1;
                        break;
                    }

                    case sf::Keyboard::S:
                    {
                        G::Camera::move_offset.z -= G::Camera::move_speed * 1;
                        break;
                    }

                    case sf::Keyboard::Space:
                    {
                        G::Camera::move_offset.y -= G::Camera::move_speed;
                        break;
                    }

                    case sf::Keyboard::LControl:
                    {
                        G::Camera::move_offset.y += G::Camera::move_speed;
                        break;
                    }
                    #endif // INTERACTIVE_MODE

                    default:
                    { break; }
                    }
                }

                default:
                { break; }
                }
            }

            // DISPLAY CONTROLLER //
            if (display_controller) continue;


            window.clear();
            {
                render.RENDER();
            }
            window.display();

            // Sleep(1000);


            if (render.test_is_finished()) { exiting_app; }
        }

    #endif

    return 0;
}
