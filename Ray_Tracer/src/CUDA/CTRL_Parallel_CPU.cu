#include "CTRL_Parallel_CPU.h"
#include "RT_Renderer.h"

#ifdef WIN
#define num_of_colors 5
Color color_tab[num_of_colors] =
{
    Color(197,64,64),
    Color(197,141,54),
    Color(101,195,54),
    Color(50,193,190),
    Color(72,61,190)
};
#endif // WIN

#ifdef CPU

// PREPARATIONS
vector<Parallel_CPU::loop_info_per_thread> Parallel_CPU::prepare_subsets_for_threads_solid_blocks(u64 num_of_threads, u64 array_size)
{
    vector<loop_info_per_thread> ret;
    u64 inc = 1;

    u64 block_size = array_size / num_of_threads;
    u64 i;
    for (i = 0; i < (num_of_threads - 1); i++)
    {
        u64 start = i * block_size;
        u64 end = (i + 1) * block_size - 1;
        ret.emplace_back(start, end, inc);
    }

    // LAST SET
    u64 start = i * block_size;
    u64 end = array_size - 1;
    ret.emplace_back(start, end, inc);

    return ret;
}
vector<Parallel_CPU::loop_info_per_thread> Parallel_CPU::prepare_subsets_for_threads_subsequent(u64 num_of_threads, u64 array_size)
{
    vector<loop_info_per_thread> ret;
    u64 end = array_size - 1;
    u64 inc = num_of_threads;

    for (u64 i = 0; i < num_of_threads; i++)
    {
        u64 start = i;
        ret.emplace_back(start, end, inc);
    }

    return ret;
}
vector<Parallel_CPU::loop_info_per_thread> Parallel_CPU::prepare_subsets_for_threads_subsequent_blocks(u64 num_of_threads, u64 block_size, u64 array_size)
{
    vector<loop_info_per_thread> ret;
    u64 end = array_size - 1;
    u64 inc = num_of_threads * block_size;

    for (u64 i = 0; i < num_of_threads; i++)
    {
        u64 start = i * block_size;
        ret.emplace_back(start, end, inc, block_size);
    }

    return ret;
}

// calling -> per pixel

#define func_call(x) ptr->per_pixel(x, G::Render::current_scene, ptr->get_my_pixel(), G::Render::current_scene->get_lights_ptr(), G::Render::current_scene->get_spheres_ptr(), G::Render::current_scene->get_details_ptr());
//#define func_call(x) ptr->set_pixel_hack_for_showing_schema(x, color_tab[thread_id % num_of_colors]);

void Parallel_CPU::thread_job_simple_schema(const Parallel_CPU::loop_info_per_thread& info, int thread_id)
{
    for (u64 i = info.start; i <= info.end; i += info.inc)
    {
        func_call(i);
    }
}
void Parallel_CPU::thread_job_complex_schema(const Parallel_CPU::loop_info_per_thread& info, int thread_id)
{
    for (u64 i = info.start; i <= info.end; i += info.inc)
    {
        if (info.end <= i + info.block_size)
        {
            for (u64 j = i; j <= info.end; j++)
            {
                func_call(j);
            }
            return;
        }

        for (u64 j = i; j < i + info.block_size; j++)
        {
            func_call(j);
        }
    }
}



Parallel_CPU::Parallel_CPU(Renderer* _ptr)
    :c_init(ptr)
{
}

// MAIN FUNCTION
void Parallel_CPU::thread_group_host_and_round_keeper(int round_counter, const int num_of_threads, u16 parallel_schema, int custom_block_size, work_group& stats)
{
    round_counter++;
    const int all_rounds = round_counter;
    u64 array_size = G::WIDTH * G::HEIGHT;

    Timer t_whole;

    // SYNC function executed by one thread
    auto on_completion = [&]() noexcept
    {
        // skip first one, nothing to stop
        if (all_rounds != round_counter)
        {
            t_whole.end();
            stats.push_whole(t_whole.get_all_in_nano());
        }

        // NEXT ROUND
        round_counter--;

        // don't start on the last one
        if (round_counter != 0)
        {
            t_whole.start();
        }
    };

    // SYNC
    std::barrier sync_point(num_of_threads, on_completion);

    // Per thread timer or just group
    #ifdef def_STATS_PER_THREAD
    #define work(x) \
    Timer t; \
    for (; round_counter != 1;) \
    { \
        sync_point.arrive_and_wait(); \
        t.start(); \
        x(info, thread_id); \
        t.end(); \
        stats.push_single(t.get_all_in_nano(), thread_id); \
    } \
    sync_point.arrive_and_wait();
    #else
    #define work(x) \
    for (; round_counter != 1;) \
    { \
        sync_point.arrive_and_wait(); \
        x(info, thread_id); \
    } \
    sync_point.arrive_and_wait();
    #endif

    // creating functions for both schemas
    auto work_comp = [&](int thread_id, const loop_info_per_thread& info)
    {
        work(thread_job_complex_schema);
    };
    auto work_simple = [&](int thread_id, const loop_info_per_thread& info)
    {
        work(thread_job_simple_schema);
    };

    vector<jthread> threads;
    threads.reserve(num_of_threads);

    // starting threads with correct work call by schema
    vector<loop_info_per_thread> schema;
    if (parallel_schema == PR_solid_blocks || parallel_schema == PR_subsequent)
    {
        if (parallel_schema == PR_solid_blocks) schema = prepare_subsets_for_threads_solid_blocks(num_of_threads, array_size);
        else schema = prepare_subsets_for_threads_subsequent(num_of_threads, array_size);
        

        for (int i = 0; i < num_of_threads; i++) { threads.emplace_back(work_simple, i, schema[i]); }
    }
    else
    {
        schema = prepare_subsets_for_threads_subsequent_blocks(num_of_threads, custom_block_size, array_size);

        for (int i = 0; i < num_of_threads; i++) { threads.emplace_back(work_comp, i, schema[i]); }
    }
}

#endif
