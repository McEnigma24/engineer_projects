#include "CTRL_Bmp.h"


void BMP_static::save(const string& file_name, const vector<RGB>& my_pixel)
{
    u32 width = def_WIDTH;
    u32 height = def_HEIGHT;

	ofstream f;
	f.open(file_name.c_str(), std::ios::out | std::ios::binary);	

	if (f.fail())
	{
		std::cout << "BMP_img could not open file\n";
		return;
	}

	unsigned char padding[3] = { 0, 0, 0 };
	const int size_padding = ((4 - (width * 3) % 4) % 4);

	const int size_header = 14;
	const int size_info_header = 40;
	const int size_file = size_header + size_info_header
						+ (3 * width * height)
						+ (size_padding * height);

	unsigned char header[size_header] {};
	unsigned char info_header[size_info_header] {};


	// HEADER //

	// Symbol
	header[0] = 'B';
	header[1] = 'M';
	// File size
	header[2] = size_file;
	header[3] = size_file >> 8;
	header[4] = size_file >> 16;
	header[5] = size_file >> 24;
	// Reserved
	header[6] = 0;
	header[7] = 0;
	header[8] = 0;
	header[9] = 0;
	// Data offset
	header[10] = size_header + size_info_header;
	header[11] = 0;
	header[12] = 0;
	header[13] = 0;


	// INFO HEADER //

	// Header size
	info_header[0] = size_info_header;
	info_header[1] = 0;
	info_header[2] = 0;
	info_header[3] = 0;
	// Width
	info_header[4] = width;
	info_header[5] = width >> 8;
	info_header[6] = width >> 16;
	info_header[7] = width >> 24;
	// Height
	info_header[8] = height;
	info_header[9] = height >> 8;
	info_header[10] = height >> 16;
	info_header[11] = height >> 24;
	// Planes
	info_header[12] = 1;
	info_header[13] = 0;
	// Bits per pixel
	info_header[14] = 24;
	info_header[15] = 0;

	f.write(reinterpret_cast<char*>(header), size_header);
	f.write(reinterpret_cast<char*>(info_header), size_info_header);

	//for (int y = 0; y < height; y++)
	for (int y = height-1; y >= 0; y--)
	{
		for (int x = 0; x < width; x++)
		{
            RGB current_color = my_pixel[def_convert_2d_to_1d(x, y)];

			unsigned char COLOR[] =
            {
                current_color.b,
                current_color.g,
                current_color.r
            };
			f.write(reinterpret_cast<char*>(COLOR), 3);
		}

		f.write(reinterpret_cast<char*>(padding), size_padding);
	}

	f.close();
}