#include "RT_Dimensions.h"

#ifdef CPU
void d1::print(const string& var_name) const
{
	print_var_name;
	var(x);
}

void d2::print(const string& var_name) const
{
	print_var_name;
	varr(x);
	var(y);
}

void d3::print(const string& var_name) const
{
	print_var_name;
	varr(x);
	varr(y);
	var(z);
}
#endif
