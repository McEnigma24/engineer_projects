#include "hip/hip_runtime.h"
#include "vec_multi.h"


#define REAL_TASK(a, b, c) \
{ \
c[i] = a[i] * b[i]; \
}


#ifdef GPU
__global__ void vec_multi_Kernel(unit* dev_a, unit* dev_b, unit* dev_c)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    REAL_TASK(dev_a, dev_b, dev_c)
}
#endif


void vec_multi::cpu_task(int i)
{
    REAL_TASK(a, b, c)
}

#ifdef GPU

    void vec_multi::cuda_prep()
    {
        // Alokacja pamięci na urządzeniu
        CCE(hipMalloc((void**)&dev_a, MEM_SIZE));
        CCE(hipMalloc((void**)&dev_b, MEM_SIZE));
        CCE(hipMalloc((void**)&dev_c, MEM_SIZE));
        
        // Kopiowanie danych do urządzenia
        CCE(hipMemcpy(dev_a, a.data(), MEM_SIZE, hipMemcpyHostToDevice));
        CCE(hipMemcpy(dev_b, b.data(), MEM_SIZE, hipMemcpyHostToDevice));
    }

    void vec_multi::cuda_launch()
    {
        vec_multi_Kernel<<<NUMBER_OF_BLOCKS, BLOCK_SIZE>>>(dev_a, dev_b, dev_c);
        CCE(hipDeviceSynchronize());
    }

    void vec_multi::cuda_finish()
    {
        // Kopiowanie wyników z powrotem na hosta
        CCE(hipMemcpy(c.data(), dev_c, MEM_SIZE, hipMemcpyDeviceToHost));

        //CCE(hipDeviceReset());
            
        CCE(hipFree(dev_a));
        CCE(hipFree(dev_b));
        CCE(hipFree(dev_c));
    }
#endif

vec_multi::vec_multi(string name, u64 s) :Benchmark_BASE(name, s, s * sizeof(unit)), a(ARRAY_SIZE), b(ARRAY_SIZE), c(ARRAY_SIZE)
{
    std::memset(a.data(), 0, MEM_SIZE);
    std::memset(b.data(), 0, MEM_SIZE);
    std::memset(c.data(), 0, MEM_SIZE);

    for(u64 i=0; i<ARRAY_SIZE; i++)
    {
        a[i] = Random_functions::random_unit_minus_1_to_1();
        b[i] = Random_functions::random_unit_minus_1_to_1();
    }
}
