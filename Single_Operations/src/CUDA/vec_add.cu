#include "hip/hip_runtime.h"
#include "vec_add.h"


#define REAL_TASK(a, b, c) \
{ \
c[i] = a[i] + b[i]; \
}


#ifdef GPU
__global__ void vec_add_Kernel(unit* dev_a, unit* dev_b, unit* dev_c)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    REAL_TASK(dev_a, dev_b, dev_c)
}
#endif


void vec_add::cpu_task(int i)
{
    REAL_TASK(a, b, c)
}

#ifdef GPU

    void vec_add::cuda_prep()
    {
        // Alokacja pamięci na urządzeniu
        CCE(hipMalloc((void**)&dev_a, MEM_SIZE));
        CCE(hipMalloc((void**)&dev_b, MEM_SIZE));
        CCE(hipMalloc((void**)&dev_c, MEM_SIZE));
        
        // Kopiowanie danych do urządzenia
        CCE(hipMemcpy(dev_a, a.data(), MEM_SIZE, hipMemcpyHostToDevice));
        CCE(hipMemcpy(dev_b, b.data(), MEM_SIZE, hipMemcpyHostToDevice));
    }

    void vec_add::cuda_launch()
    {
        vec_add_Kernel<<<NUMBER_OF_BLOCKS, BLOCK_SIZE>>>(dev_a, dev_b, dev_c);
        CCE(hipDeviceSynchronize());
    }

    void vec_add::cuda_finish()
    {
        // Kopiowanie wyników z powrotem na hosta
        CCE(hipMemcpy(c.data(), dev_c, MEM_SIZE, hipMemcpyDeviceToHost));

        //CCE(hipDeviceReset());
            
        CCE(hipFree(dev_a));
        CCE(hipFree(dev_b));
        CCE(hipFree(dev_c));
    }
#endif

vec_add::vec_add(string name, u64 s) :Benchmark_BASE(name, s, s * sizeof(unit)), a(ARRAY_SIZE), b(ARRAY_SIZE), c(ARRAY_SIZE)
{
    std::memset(a.data(), 0, MEM_SIZE);
    std::memset(b.data(), 0, MEM_SIZE);
    std::memset(c.data(), 0, MEM_SIZE);

    for(u64 i=0; i<ARRAY_SIZE; i++)
    {
        a[i] = Random_functions::random_unit_minus_1_to_1();
        b[i] = Random_functions::random_unit_minus_1_to_1();
    }
}
