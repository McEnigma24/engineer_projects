#include "hip/hip_runtime.h"
#include "benchmark_base.h"
#include <sstream>
#include "_SINGLE_.h"
#include <iomanip>

// COMMON //

#define PROGRESS_LOG

#ifdef PROGRESS_LOG
#define OLD_LOG_LINE(...)  
#define PROGRESS_LOG_LINE(...) __VA_ARGS__
#else
#define OLD_LOG_LINE(...) __VA_ARGS__
#define PROGRESS_LOG_LINE(...) 
#endif

#ifdef CPU
void Benchmark_BASE::omp_launch()
{
    #pragma omp parallel for schedule(static)
    for(int i=0; i<ARRAY_SIZE; i++)
    {
        cpu_task(i);
    }
}
#endif

Benchmark_BASE::Benchmark_BASE(string dn, u64 array_size, u64 mem_size)
    :derived_name(dn), ARRAY_SIZE(array_size), MEM_SIZE(mem_size),
    CPU_LINE(all_stats(def_MAX_NUM_OF_THREADS + 1))
    GPU_LINE(all_stats(1))
{
    GPU_LINE(setup_block_size());
}

#ifdef GPU
void Benchmark_BASE::setup_block_size()
{
    // <<<NUMBER_OF_BLOCKS, BLOCK_SIZE>>>

    // BLOCK_SIZE = ARRAY_SIZE; // no difference
    // NUMBER_OF_BLOCKS = 1;

    // BLOCK_SIZE = 1; // może warto spróbować          // JEST duża różnica i jest turbo WOLNO
    // NUMBER_OF_BLOCKS = ARRAY_SIZE;

    BLOCK_SIZE = def_BLOCK_SIZE;
    NUMBER_OF_BLOCKS = ARRAY_SIZE / BLOCK_SIZE + 1; // jest +1 żeby pokrywało wszystko -> przez to musi być warunek
}
#endif


float calculate_progress()
{
    float ret = (((float) Global::progress_counter) / ((float) Global::progress_all));

    if(!((Global::scaling_multi == -1) && (Global::scaling_add == -1)))
    {
        ret = ret * Global::scaling_multi + Global::scaling_add;
    }
    else
    {
        // DEFAULT //
        ret = ret * 1.0f + 0.0f;
    }

    return ret;
}

string get_current_local_time()
{
    ostringstream oss;

    if(Global::REP_NUMBER == 1)
    {
        auto now = std::chrono::system_clock::now();
        std::time_t czas = std::chrono::system_clock::to_time_t(now);

        // Korzystamy z std::localtime w systemach Linux/Unix
        std::tm* aktualnyCzas = std::localtime(&czas);  // Zwraca wskaźnik na strukturę tm

        oss << std::put_time(aktualnyCzas, "%H:%M:%S") << std::endl;
    }
    else
    {
        // Pobranie aktualnego czasu lokalnego
        auto now = std::chrono::system_clock::now();
        std::time_t current_time = std::chrono::system_clock::to_time_t(now);
        std::tm* current_tm = std::localtime(&current_time);

        // Odczyt daty z pliku
        std::ifstream file("../_run_time_config_/estimated_finish_date.txt");
        if (!file.is_open()) {
            std::cerr << "Nie można otworzyć pliku: " << "../_run_time_config_/estimated_finish_date.txt" << std::endl;
            exit(0);
        }

        int year, month, day, hour, minute, second; // 2024 11 15 10 48 02
        file >> year >> month >> day >> hour >> minute >> second;
        file.close();

        // cout << year << " " << month << " " << day << " " << hour << " " << minute << " " << second << "   ";

        // Stworzenie obiektu tm dla daty z pliku
        std::tm target_tm = {};
        target_tm.tm_year = year - 1900;  // tm_year jest liczbą lat od 1900
        target_tm.tm_mon = month - 1;    // tm_mon jest liczbą miesięcy od 0 (styczeń)
        target_tm.tm_mday = day;
        target_tm.tm_hour = hour;
        target_tm.tm_min = minute;
        target_tm.tm_sec = second;

        // Konwersja na std::time_t
        std::time_t target_time = std::mktime(&target_tm);
        if (target_time == -1) {
            std::cerr << "Nieprawidłowa data w pliku." << std::endl;
            return "error";
        }

        // Obliczenie różnicy czasu
        //auto diff = std::difftime(current_time, target_time);
        auto diff = std::difftime(target_time, current_time);
        if (diff < 0) {
            // std::cout << "Podana data już minęła." << std::endl;
            return "very close to finishing...\n";
        }

        // Przekształcenie różnicy na lata, miesiące, dni, godziny, minuty i sekundy
        auto diff_seconds = static_cast<int>(diff);
        int days = diff_seconds / (24 * 3600);
        diff_seconds %= (24 * 3600);
        int hours = diff_seconds / 3600;
        diff_seconds %= 3600;
        int minutes = diff_seconds / 60;
        int seconds = diff_seconds % 60;

        // Wyświetlenie wyniku

        if(days != 0) oss << days << "d ";
        if(hours != 0) oss << hours << "h ";
        if(minutes != 0) oss << minutes << "m ";
        if(seconds != 0) oss << seconds << "s";
        oss << "\n";
    }

	return oss.str();
}

void Benchmark_BASE::single_round(int num_of_threads)
{
    OLD_LOG_LINE(cout << "single round " << num_of_threads << "\n");

    PROGRESS_LOG_LINE(printf("[%.0f%]", (100.0f * calculate_progress())));
    PROGRESS_LOG_LINE(cout << "       " << get_current_local_time();)
    GPU_LINE(cuda_prep());
    GPU_LINE(CCE(hipDeviceSynchronize()));
    CPU_LINE(omp_set_num_threads(num_of_threads););

    Timer t;

    t.start();
    {
        CPU_LINE(omp_launch());
        GPU_LINE(cuda_launch());
    }
    t.end();
    PROGRESS_LOG_LINE(Global::progress_counter++);

    all_stats[num_of_threads].push(t.get_all_in_nano());

    GPU_LINE(cuda_finish());
    //GPU_LINE(hipDeviceReset());
}

void Benchmark_BASE::round_controller()
{
    OLD_LOG_LINE(cout << "round controller\n");
    for(int rep = 0; rep < Global::REP_NUMBER; rep++)
    {
        OLD_LOG_LINE(cout << rep + 1 << endl);
        
        CPU_LINE(for(int i=1; i <= def_MAX_NUM_OF_THREADS; i++) { single_round(i); });

        GPU_LINE(single_round(0));
    }
}

/////////////////////////////////////////////////////////////////////////////

#include "_PARSER_Format_Buffer_Common_IMP_.h"

/////////////////////////////////////////////////////////////////////////////

void Benchmark_BASE::save_stats()
{
    #ifdef CPU
    stringstream stream;
    stream << fixed << std::setprecision(2) << (float)omp_get_num_procs() / (float)stoi(Global::num_of_cores);
    #endif

    tupl variables =
    {
        0.0,
        0,
        0,
        0,

        CPU_LINE("CPU")
        GPU_LINE("GPU")
        ,

        Global::model_name,

        ARCH_X86_LINE("x86")
        ARCH_X64_LINE("x64")
        GPU_LINE("CUDA")
        ,

        UNIT_FLOAT_LINE("float")
        UNIT_DOUBLE_LINE("double")
        ,

        Global::num_of_cores,

        CPU_LINE(to_string(omp_get_num_procs()))
        GPU_LINE(to_string(0))
        ,

        CPU_LINE(stream.str())
        GPU_LINE(to_string(0))
        ,

        derived_name,
        to_string(ARRAY_SIZE),
        
        CPU_LINE(to_string(0))
        GPU_LINE(to_string(0))
        ,
        
        CPU_LINE("measuring")
        GPU_LINE("BEST")
    };

    #ifdef CPU
        tupl best_time_variables;
        vector<u64> best_data;
        u64 best_time_so_far = -1; // biggest uint64_t

        for(int i=1; i <= def_MAX_NUM_OF_THREADS; i++)
        {
            auto [min, avg, max, rel_dev] = all_stats[i].get();

            GET(variables, 0) = rel_dev;

            GET(variables, NUM_OF_THREADS + 4) = to_string(i);
            GET(variables, 1) = min;
            GET(variables, 2) = avg;
            GET(variables, 3) = max;

            if(avg < best_time_so_far)
            {
                best_data = all_stats[i].get_data();
                best_time_so_far = avg;
                best_time_variables = variables;
            }

            string output = Format_Buffer::input_variables_return_log_line(variables);
            insert_raw_datapoints(output, all_stats[i].get_data());
            output += "\n";
            OUTPUT_TO_FILE(common_file, output);
        }
        GET(best_time_variables, RATING + 4) = "BEST";
        string output = Format_Buffer::input_variables_return_log_line(best_time_variables);
        insert_raw_datapoints(output, best_data);
        output += "\n";
        OUTPUT_TO_FILE(common_file, output);
    #endif

    #ifdef GPU
        auto [min, avg, max, rel_dev] = all_stats[0].get();

        GET(variables, 0) = rel_dev;

        GET(variables, 1) = min;
        GET(variables, 2) = avg;
        GET(variables, 3) = max;

        string output = Format_Buffer::input_variables_return_log_line(variables);
        insert_raw_datapoints(output, all_stats[0].get_data());
        output += "\n";
        OUTPUT_TO_FILE(common_file, output);
    #endif
}

void Benchmark_BASE::start()
{
    OLD_LOG_LINE(cout << "starting -> " << derived_name << " ( " << ARRAY_SIZE << " )" << endl);

    round_controller();
    save_stats();
}

/////////////////////////////////////////////////////////

// CUSTOM for any derived //

CPU_LINE(inline void Benchmark_BASE::cpu_task(int i) { FATAL("virtual function not implemented"); })

GPU_LINE(void Benchmark_BASE::cuda_prep() { FATAL("virtual function not implemented"); })
GPU_LINE(void Benchmark_BASE::cuda_launch() { FATAL("virtual function not implemented"); })
GPU_LINE(void Benchmark_BASE::cuda_finish() { FATAL("virtual function not implemented"); })

// + KONSTRUKTOR gdzie przyjmuje jakieś dane i robi ich prep + ustawiał ARRAY_SIZE